
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define GET_TIME(X, Y) (((Y).tv_sec - (X).tv_sec) + ((Y).tv_nsec - (X).tv_nsec) / 1000000000.0)

   __device__ int IE_d;
   __device__ int JE_d;
   __device__ float cb_d;

  __global__ void ezCalc ( float *ez, float *hx, float *hy ) {
    //float cb = 133.105;
    //int IE = 10, JE = 10;
    int i = threadIdx.x, j = blockIdx.x;

    if (j == 0) { // at x=0
      if (i == 0 || i == IE_d - 1) // at x=0,y=0
        ez[j * IE_d + i] = 0.0;
      else
        ez[j * IE_d + i] = ez[j * IE_d + i] + cb_d * (hy[j * IE_d + i] - hy[j * IE_d + (i - 1)] + hx[(j - 1 + JE_d) * IE_d + i] - hx[j * IE_d + i]);
    } else {
      if (i == 0 || i == IE_d - 1)
        ez[j * IE_d + i] = 0.0;
      else
        ez[j * IE_d + i] = ez[j * IE_d + i] + cb_d * (hy[j * IE_d + i] - hy[j * IE_d + (i - 1)] + hx[(j - 1) * IE_d + i] - hx[j * IE_d + i]);
    }

  }

  int main(int argc, char * argv[]) {

    int IE, JE, nsteps;
    int i, j, n, is, jc, y, k;
    int xstart;
    float pi = 3.141592653589793238462643;
    float * ez, * hx, * hy;
    float * ez_d, *hx_d, * hy_d;
    float dx, dt, epsz, mu, courant, cb, db, c, x, t, lambda, freq;
    int tStart, tTotal;
    int size;

    FILE * fp;

    if (argc != 4) {
      printf("Invalid arguments... please type:\n");
      printf("   %s  IE JE steps\n", argv[0]);
      exit(0);
    }

    IE = atoi(argv[1]);
    JE = atoi(argv[2]);
    nsteps = atoi(argv[3]);

    printf("Running 2D FDTD algorithm with matrix of size %d x %d (%d steps)\n", IE, JE, nsteps);

    struct timespec Begin, Step1, Step2, End;
    double diff, accum;

    is = 10;

    epsz = 8.854e-12;
    mu = 4.0 * pi * 1.0e-7;
    c = 3.0e8;

    courant = 0.5;
    dx = 0.001;
    dt = (courant * dx) / (sqrt(2) * c);

    cb = dt / (epsz * dx);
    db = dt / (mu * dx);

    printf("Coefficients are: dt=%g cb=%g db=%g\n", dt, cb, db);

    size = IE * JE;

    ez = (float * ) calloc(size, sizeof(float));
    hx = (float * ) calloc(size, sizeof(float));
    hy = (float * ) calloc(size, sizeof(float));

    hipMalloc( (void **) &ez_d, size * sizeof(float));
    hipMalloc( (void **) &hx_d, size * sizeof(float));
    hipMalloc( (void **) &hy_d, size * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(cb_d), &cb, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(JE_d), &JE, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(IE_d), &IE, sizeof(float), 0, hipMemcpyHostToDevice);

    freq = 50e9;

    accum = 0.0;
    for (n = 0; n < nsteps; n++) { // TIME
      if (clock_gettime(CLOCK_REALTIME, &Begin) == -1) {
        perror("Error in gettime");
        exit(1);
      }

      hipMemcpy( ez_d, ez, size * sizeof(float), hipMemcpyHostToDevice );
      hipMemcpy( hx_d, hx, size * sizeof(float), hipMemcpyHostToDevice );
      hipMemcpy( hy_d, hy, size * sizeof(float), hipMemcpyHostToDevice );

      //Calculate the Ez field
      ezCalc<<<JE, IE>>>( ez_d, hx_d, hy_d );

      hipMemcpy( ez, ez_d, size * sizeof(float), hipMemcpyDeviceToHost );

      clock_gettime(CLOCK_REALTIME, &Step1);

      for (j = 0; j < JE; j++) { // x dimension
        ez[j * IE + is] = cos(2 * pi * freq * n * dt);
      }

      clock_gettime(CLOCK_REALTIME, &Step2);

      //Calculate the H field
      for (j = 0; j < JE; j++) {
        for (i = 0; i < IE; i++) { // y dimension
          // Periodic Boundary implementation
          if (j + 1 == JE)
            hx[j * IE + i] = hx[j * IE + i] + db * (ez[j * IE + i] - ez[i]);
          else
            hx[j * IE + i] = hx[j * IE + i] + db * (ez[j * IE + i] - ez[(j + 1) * JE + i]);

          if (i == IE - 1)
            hy[j * JE + i] = hy[j * JE + i] + db * (0 - ez[j * JE + i]);
          else
            hy[j * JE + i] = hy[j * JE + i] + db * (ez[j * JE + (i + 1)] - ez[j * JE + i]);
        }
      } //Hx calculation

      if (clock_gettime(CLOCK_REALTIME, &End) == -1) {
        perror("Error in gettime");
        exit(1);
      }

      diff = GET_TIME(Begin, End);
      accum += diff;

      printf("\n====Iteration (%d)====", n);
      printf("\n======Total time: (%f)====", diff);
      printf("\n======Part 1: (%f)====", GET_TIME(Begin, Step1));
      printf("\n======Part 2: (%f)====", GET_TIME(Step1, Step2));
      printf("\n======Part 3: (%f)====", GET_TIME(Step2, End));
    }

    printf("\n\n====Total time: %f\n", accum);

    // write output to file
    fp = fopen("output_gpu_v2.txt", "w");
    fprintf(fp, "==================== Ez MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", ez[i]);
    }
    fprintf(fp, "==================== Hx MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hx[i]);
    }
    fprintf(fp, "==================== Hy MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hy[i]);
    }

    free(ez);
    free(hy);
    free(hx);

    hipFree( ez_d );
    hipFree( hx_d );
    hipFree( hy_d );

    return 0;

  }
