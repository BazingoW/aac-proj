
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define GET_TIME(X, Y) (((Y).tv_sec - (X).tv_sec) + ((Y).tv_nsec - (X).tv_nsec) / 1000000000.0)
#define THREADS_PER_BLOCK 1024

  __constant__ __device__ int IE_d;
  __constant__ __device__ int JE_d;
  __constant__ __device__ float cb_d;
  __constant__ __device__ int is_d;
  __constant__ __device__ float pi_d;
  __constant__ __device__ float freq_d;
  __constant__ __device__ float dt_d;
  __constant__ __device__ float db_d;

  __global__ void ezCalc ( float *ez, float *hx, float *hy ) {
    int i, j = blockIdx.x;

    for (i = threadIdx.x; i < IE_d; i += blockDim.x) {

      if (i == 0 || i == IE_d - 1)
        ez[j * IE_d + i] = 0.0;

      if (j != 0 && !(i == 0 || i == IE_d - 1)) { // at x=0
        ez[j * IE_d + i] = ez[j * IE_d + i] + cb_d * (hy[j * IE_d + i] - hy[j * IE_d + (i - 1)] + hx[(j - 1 + JE_d) * IE_d + i] - hx[j * IE_d + i]);
      } else if (!(i == 0 || i == IE_d - 1))
        ez[j * IE_d + i] = ez[j * IE_d + i] + cb_d * (hy[j * IE_d + i] - hy[j * IE_d + (i - 1)] + hx[(j - 1) * IE_d + i] - hx[j * IE_d + i]);
      }
    }

  

  __global__ void ezCalc2 ( float *ez , int n ) {
    int j;

    for (j = threadIdx.x; j < JE_d; j += blockDim.x) {
      ez[j * IE_d + is_d] = cos(2 * pi_d * freq_d * n * dt_d);
    }

  }

  __global__ void hCalc ( float *ez, float *hx, float *hy ) {
    int i, j = blockIdx.x;

    for (i = threadIdx.x; i < IE_d; i += blockDim.x) {
      if (j + 1 == JE_d)
        hx[j * IE_d + i] = hx[j * IE_d + i] + db_d * (ez[j * IE_d + i] - ez[i]);
      else
        hx[j * IE_d + i] = hx[j * IE_d + i] + db_d * (ez[j * IE_d + i] - ez[(j + 1) * JE_d + i]);

      if (i == IE_d - 1)
        hy[j * JE_d + i] = hy[j * JE_d + i] + db_d * (0 - ez[j * JE_d + i]);
      else
        hy[j * JE_d + i] = hy[j * JE_d + i] + db_d * (ez[j * JE_d + (i + 1)] - ez[j * JE_d + i]);
    }


  }

  int main(int argc, char * argv[]) {

    int IE, JE, nsteps;
    int i, j, n, is;
    float pi = 3.141592653589793238462643;
    float * ez, * hx, * hy;
    float * ez_d, *hx_d, * hy_d;
    float dx, dt, epsz, mu, courant, cb, db, c, freq;
    int size;
    struct timespec Begin, Step0, Step1, Step2, Step3, End;
    FILE * fp;

    if (argc != 4) {
      printf("Invalid arguments... please type:\n");
      printf("   %s  IE JE steps\n", argv[0]);
      exit(0);
    }

    IE = atoi(argv[1]);
    JE = atoi(argv[2]);
    nsteps = atoi(argv[3]);

    printf("Running 2D FDTD algorithm with matrix of size %d x %d (%d steps)\n", IE, JE, nsteps);

    hipMemcpyToSymbol(HIP_SYMBOL(pi_d), &pi, sizeof(float), 0, hipMemcpyHostToDevice);

    is = 10;
    hipMemcpyToSymbol(HIP_SYMBOL(is_d), &is, sizeof(int), 0, hipMemcpyHostToDevice);

    epsz = 8.854e-12;
    mu = 4.0 * pi * 1.0e-7;
    c = 3.0e8;

    courant = 0.5;
    dx = 0.001;
    dt = (courant * dx) / (sqrt(2) * c);
    hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &dt, sizeof(float), 0, hipMemcpyHostToDevice);

    cb = dt / (epsz * dx);
    db = dt / (mu * dx);
    hipMemcpyToSymbol(HIP_SYMBOL(cb_d), &cb, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(db_d), &db, sizeof(float), 0, hipMemcpyHostToDevice);

    printf("Coefficients are: dt=%g cb=%g db=%g\n", dt, cb, db);

    size = IE * JE;

    ez = (float * ) calloc(size, sizeof(float));
    hx = (float * ) calloc(size, sizeof(float));
    hy = (float * ) calloc(size, sizeof(float));

    hipMalloc( (void **) &ez_d, size * sizeof(float));
    hipMalloc( (void **) &hx_d, size * sizeof(float));
    hipMalloc( (void **) &hy_d, size * sizeof(float));

    freq = 50e9;
    hipMemcpyToSymbol(HIP_SYMBOL(freq_d), &freq, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(JE_d), &JE, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(IE_d), &IE, sizeof(float), 0, hipMemcpyHostToDevice);

    if (clock_gettime(CLOCK_REALTIME, &Begin) == -1) {
      perror("Error in gettime");
      exit(1);
    }

    // Transfer initial matrices to gpu
    hipMemcpy( ez_d, ez, size * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( hx_d, hx, size * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( hy_d, hy, size * sizeof(float), hipMemcpyHostToDevice );

    for (n = 0; n < nsteps; n++) { // TIME
      if (clock_gettime(CLOCK_REALTIME, &Step0) == -1) {
        perror("Error in gettime");
        exit(1);
      }

      //Calculate the Ez field
      ezCalc<<<JE, THREADS_PER_BLOCK>>>( ez_d, hx_d, hy_d );

      clock_gettime(CLOCK_REALTIME, &Step1);

      //Ez field generator (line)
      ezCalc2<<<1, THREADS_PER_BLOCK>>>( ez_d , n );

      clock_gettime(CLOCK_REALTIME, &Step2);

      //Calculate the H field
      hCalc<<<JE, THREADS_PER_BLOCK>>>( ez_d, hx_d, hy_d );

      if (clock_gettime(CLOCK_REALTIME, &Step3) == -1) {
        perror("Error in gettime");
        exit(1);
      }

    }

    // Retrieve matrices from gpu
    hipMemcpy( ez, ez_d, size * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( hx, hx_d, size * sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy( hy, hy_d, size * sizeof(float), hipMemcpyDeviceToHost );

    if (clock_gettime(CLOCK_REALTIME, &End) == -1) {
      perror("Error in gettime");
      exit(1);
    }

    printf("\n\n====Total time: %f\n", GET_TIME(Begin, End));

    // write output to file
    fp = fopen("output_gpu_v4.txt", "w");
    fprintf(fp, "==================== Ez MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", ez[i]);
    }
    fprintf(fp, "==================== Hx MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hx[i]);
    }
    fprintf(fp, "==================== Hy MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hy[i]);
    }

    free(ez);
    free(hy);
    free(hx);

    hipFree( ez_d );
    hipFree( hx_d );
    hipFree( hy_d );

    return 0;

  }
