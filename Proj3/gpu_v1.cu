#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define GET_TIME(X, Y) (((Y).tv_sec - (X).tv_sec) + ((Y).tv_nsec - (X).tv_nsec) / 1000000000.0)

  __constant__ float cb_d;

  __global__ void ezCalc(float *ez, float *hx, float *hy) {
    int IE = blockDim.x;
    int JE = IE;
    int i = threadIdx.x;
    int j = blockIdx.x;
    float cb = cb_d;
    //int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (j == 0) { // at x=0
      if (i == 0 || i == IE - 1) // at x=0,y=0
        ez[j * IE + i] = 0.0;
      else
        ez[j * IE + i] = ez[j * IE + i] + cb * (hy[j * IE + i] - hy[j * IE + (i - 1)] + hx[(j - 1 + JE) * IE + i] - hx[j * IE + i]);
    } else if (j < JE) {
      if (i == 0 || i == IE - 1)
        ez[j * IE + i] = 0.0;
      else
        ez[j * IE + i] = ez[j * IE + i] + cb * (hy[j * IE + i] - hy[j * IE + (i - 1)] + hx[(j - 1) * IE + i] - hx[j * IE + i]);
    }
  }

  int main(int argc, char * argv[]) {

    int IE, JE, nsteps;
    int i, j, n, is, jc, y, k;
    int xstart;
    float pi = 3.141592653589793238462643;
    float * ez_h, * hx_h, * hy_h;
    float * ez_d, * hx_d, * hy_d;
    float dx, dt, epsz, mu, courant, cb, db, c, x, t, lambda, freq;
    int tStart, tTotal;

    FILE * fp;

    //ta mal provavelmoente
    dim3 block_config(IE,1,1);
    dim3 grid_config(1,JE,1);



    if (argc != 4) {
      printf("Invalid arguments... please type:\n");
      printf("   %s  IE JE steps\n", argv[0]);
      exit(0);
    }

    IE = atoi(argv[1]);
    JE = atoi(argv[2]);
    nsteps = atoi(argv[3]);

    printf("Running 2D FDTD algorithm with matrix of size %d x %d (%d steps)\n", IE, JE, nsteps);
    int size = IE * JE;

    struct timespec Begin, Step1, Step2, End;
    double diff, accum;

    is = 10;

    epsz = 8.854e-12;
    mu = 4.0 * pi * 1.0e-7;
    c = 3.0e8;

    courant = 0.5;
    dx = 0.001;
    dt = (courant * dx) / (sqrt(2) * c);

    cb = dt / (epsz * dx);
    db = dt / (mu * dx);

    printf("Coefficients are: dt=%g cb=%g db=%g\n", dt, cb, db);

    ez_h = (float * ) calloc(size, sizeof(float));
    hx_h = (float * ) calloc(size, sizeof(float));
    hy_h = (float * ) calloc(size, sizeof(float));

    // Init device and allocate memory
    if (hipSetDevice(0) != hipSuccess) { perror("CANNOT SET DEVICE"); }
    if (hipMalloc((void **) &ez_d, size * sizeof(float))) { perror("CANNOT MALLOC"); }
    if (hipMalloc((void **) &hx_d, size * sizeof(float))) { perror("CANNOT MALLOC"); }
    if (hipMalloc((void **) &hy_d, size * sizeof(float))) { perror("CANNOT MALLOC"); }

    freq = 50e9;

    hipMemcpyToSymbol(HIP_SYMBOL(cb_d), &cb, sizeof(float));

    accum = 0.0;
    for (n = 0; n < nsteps; n++) { // TIME
      if (clock_gettime(CLOCK_REALTIME, &Begin) == -1) {
        perror("Error in gettime");
        exit(1);
      }

      // Copy matrices to device
      hipMemcpy(ez_d, ez_h, size, hipMemcpyHostToDevice);
      hipMemcpy(hx_d, hx_h, size, hipMemcpyHostToDevice);
      hipMemcpy(hy_d, hy_h, size, hipMemcpyHostToDevice);

      /* Lauch ez calculation kernel
         JE blocks, one for each matrix line, and IE threads per block,
         one for each matrix element
       */

      ezCalc<<<JE, IE>>>(ez_d, hx_d, hy_d);

      // Copy ez matrix to host
      hipMemcpy(ez_h, ez_d, size, hipMemcpyDeviceToHost);

      clock_gettime(CLOCK_REALTIME, &Step1);

      for (j = 0; j < JE; j++) { // x dimension
        ez_h[j * IE + is] = cos(2 * pi * freq * n * dt);
      }

      clock_gettime(CLOCK_REALTIME, &Step2);

      //Calculate the H field
      for (j = 0; j < JE; j++) {
        for (i = 0; i < IE; i++) { // y dimension
          // Periodic Boundary implementation
          if (j + 1 == JE)
            hx_h[j * IE + i] = hx_h[j * IE + i] + db * (ez_h[j * IE + i] - ez_h[i]);
          else
            hx_h[j * IE + i] = hx_h[j * IE + i] + db * (ez_h[j * IE + i] - ez_h[(j + 1) * JE + i]);

          if (i == IE - 1)
            hy_h[j * JE + i] = hy_h[j * JE + i] + db * (0 - ez_h[j * JE + i]);
          else
            hy_h[j * JE + i] = hy_h[j * JE + i] + db * (ez_h[j * JE + (i + 1)] - ez_h[j * JE + i]);
        }
      } //Hx calculation

      if (clock_gettime(CLOCK_REALTIME, &End) == -1) {
        perror("Error in gettime");
        exit(1);
      }

      diff = GET_TIME(Begin, End);
      accum += diff;

      printf("\n====Iteration (%d)====", n);
      printf("\n======Total time: (%f)====", diff);
      printf("\n======Part 1: (%f)====", GET_TIME(Begin, Step1));
      printf("\n======Part 2: (%f)====", GET_TIME(Step1, Step2));
      printf("\n======Part 3: (%f)====", GET_TIME(Step2, End));
    }

    printf("\n\n====Total time: %f\n", accum);

    // write output to file
    fp = fopen("output_gpu_v1.txt", "w");
    fprintf(fp, "==================== Ez MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", ez_h[i]);
    }
    fprintf(fp, "==================== Hx MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hx_h[i]);
    }
    fprintf(fp, "==================== Hy MATRIX ========================\n");
    for (i = 0, j = 0;
      (i < IE * JE) && (i < 1000); i++, j++) {
      if (j == 8) {
        fprintf(fp, "\n");
        j = 0;
      }
      fprintf(fp, "%8f  ", hy_h[i]);
    }

    fclose(fp);

    // Cleanup
    free(ez_h);
    free(hy_h);
    free(hx_h);
    hipFree(ez_d);
    hipFree(hy_d);
    hipFree(hx_d);

    return 0;

  }
